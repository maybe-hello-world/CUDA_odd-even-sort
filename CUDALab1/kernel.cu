#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

// ---------------------------------
// BEGIN OF USER AREA

// Debug level, possible values: 0 - 5, 5 is highest
// Highest level will cause EXTREMELY detailed output (the whole array will be printed)
__constant__ const int DEBUG_LEVEL = 4;

// Array size for initialization, used only in inputArray functiont
__constant__ const int G_ARRAY_SIZE = 8192;

// Number of threads inside of block
__constant__ const int BLOCK_SIZE = 8;

int inputArray(int ** _arr) {
	int arr_size = G_ARRAY_SIZE;
	*_arr = new int[arr_size];

	for (int i = 0; i < arr_size; i++) {
		(*_arr)[i] = rand() % arr_size;
	}

	if (DEBUG_LEVEL >= 5) {
		std::wcout << "Array: ";
		for (int i = 0; i < arr_size; i++) {
			std::wcout << (*_arr)[i] << ", ";
		}
		std::wcout << std::endl;
	}


	return arr_size;
}

void outputArray(int * _arr, int arr_size) {
	if (DEBUG_LEVEL >= 5) {
		std::wcout << "Array: ";
		for (int i = 0; i < arr_size; i++) {
			std::wcout << _arr[i] << ", ";
		}
		std::wcout << std::endl;
	}
	
	bool sorted = true;
	for (int i = 1; i < arr_size; i++) {
		if (_arr[i] < _arr[i - 1]) {
			sorted = false;
			break;
		}
	}

	if (DEBUG_LEVEL >= 1) std::wcout << "Array sorting check, sorted: " << std::boolalpha << sorted << std::endl;
}

// END OF USER AREA
// ---------------------------------

// Number of blocks
__constant__ const int GRID_SIZE = G_ARRAY_SIZE / 2 / BLOCK_SIZE;

void pause() {
	std::wcout << "Press enter to continue . . . " << std::endl;
	std::cin.ignore();
}

bool inline cudaErrorOccured(hipError_t _cudaStatus) {
	if (_cudaStatus != hipSuccess) {
		std::wcout << std::endl << std::endl
			<< "------------------------------"
			<< "CUDA error: " << _cudaStatus << std::endl;
		if (DEBUG_LEVEL >= 1) std::wcout << hipGetErrorString(_cudaStatus) << std::endl;
		std::wcout 
			<< "------------------------------"
			<< std::endl << std::endl;

		return true;
	}
	return false;
}

__device__ bool D_SORTED = false;

__device__ inline void swap(int * arr, int i, int j) {
	int tmp = arr[i];
	arr[i] = arr[j];
	arr[j] = tmp;
}

__global__ void kernel(int * arr, int parity) {
	//get own index
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//array for swapping
	__shared__ int shared_arr[BLOCK_SIZE * 2];

	//copying forth	
	int last_deduction = 0;
	if (threadIdx.x == 0) {
		if (parity == 1 && blockIdx.x == GRID_SIZE - 1) last_deduction = 1;

		for (int i = 0; i < blockDim.x * 2 - last_deduction; i++) {
			shared_arr[i] = arr[2 * idx + i + parity];
		}
	}
	
	__syncthreads();

	// Last kernel shouldn't work in this case
	if (parity == 1 && idx == BLOCK_SIZE * GRID_SIZE - 1) return;

	//swapping
	if (shared_arr[threadIdx.x * 2] > shared_arr[threadIdx.x * 2 + 1]) {
		swap(shared_arr, threadIdx.x * 2, threadIdx.x * 2 + 1);
		D_SORTED = false;
	}

	__syncthreads();


	//copying back
	if (threadIdx.x == 0) {
		for (int i = 0; i < blockDim.x * 2 - last_deduction; i++) {
			arr[2 * idx + i + parity] = shared_arr[i];
		}
	}
}

void oddevensort(int * arr, int arr_size) {
	bool sorted = false;
	hipError_t cudaStatus = hipSuccess;
	int counter = 0;

	while (!sorted) {
		sorted = true;
		
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(D_SORTED), &sorted, sizeof(bool));
		if (cudaErrorOccured(cudaStatus)) return;

		kernel<<<GRID_SIZE, BLOCK_SIZE>>>(arr, 0);
		kernel<<<GRID_SIZE, BLOCK_SIZE>>>(arr, 1);

		cudaStatus = hipMemcpyFromSymbol(&sorted, HIP_SYMBOL(D_SORTED), sizeof(bool));
		if (cudaErrorOccured(cudaStatus)) return;
		counter++;
	}

	if (DEBUG_LEVEL >= 1) std::cout << "Sorting finished, iterations: " << counter << std::endl;
}

int main()
{
	hipError_t cudaStatus = hipSuccess;

	int arr_size = 0;
	int * arr = 0;
	int * d_arr = 0; //GPU copy of array

	//0. ��������� ���������� � CUDA device'��
	if (DEBUG_LEVEL >= 1)
	{
		std::wcout << "CUDA realization of odd-even sorting algorithm" << std::endl;
		std::wcout << "Author: Roman Beltyukov" << std::endl << std::endl;

		std::wcout << "CUDA information" << std::endl;
		int deviceCount = 0;
		cudaStatus = hipGetDeviceCount(&deviceCount);
		if (cudaErrorOccured(cudaStatus)) return 1;
		std::wcout << "Available CUDA device count: " << deviceCount << std::endl << std::endl;

		hipDeviceProp_t devProps;
		for (int i = 0; i < deviceCount; i++) {
			cudaStatus = hipGetDeviceProperties(&devProps, i);
			if (cudaErrorOccured(cudaStatus)) return 1;

			std::wcout
				<< "Device #" << i << ", CUDA version: " << devProps.major << "." << devProps.minor
				<< ", integrated: " << std::boolalpha << devProps.integrated << std::endl
				<< "Name: " << devProps.name << std::endl
				<< "Clockrate: " << (double)devProps.clockRate / 1024 << "MHz" << std::endl
				<< "Total global memory: " << (double)devProps.totalGlobalMem / 1024 / 1024 / 1024 << "GB" << std::endl
				<< "Shared memory per block: " << (double)devProps.sharedMemPerBlock / 1024 << "KB" << std::endl
				<< "Warp size: " << devProps.warpSize << std::endl
				<< "Max threads per block: " << devProps.maxThreadsPerBlock << std::endl
				<< "Max threads dimension: [" 
					<< devProps.maxThreadsDim[0] << ", " 
					<< devProps.maxThreadsDim[1] << ", " 
					<< devProps.maxThreadsDim[2] << "]" << std::endl
				<< "Max grid size: [" 
					<< devProps.maxGridSize[0] << ", " 
					<< devProps.maxGridSize[1] << ", " 
					<< devProps.maxGridSize[0] << "]" << std::endl
				<< std::endl;
		}
		std::wcout << std::endl;
	}

	//1. ��������� �������
	arr_size = inputArray(&arr);
	if (DEBUG_LEVEL >= 1) std::wcout << "Array generated, size: " << arr_size << ", last element: " << arr[arr_size - 1] << std::endl;

	//2. ������������� ������ �� ���������� � ����������� ������� ����
	cudaStatus = hipMalloc((void **)&D_SORTED, sizeof(bool));
	if (cudaErrorOccured(cudaStatus)) return 1;

	cudaStatus = hipMalloc((void **)&d_arr, arr_size * sizeof(int));
	if (cudaErrorOccured(cudaStatus)) return 1;

	cudaStatus = hipMemcpy(d_arr, arr, arr_size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaErrorOccured(cudaStatus)) return 1;

	if (DEBUG_LEVEL >= 1) std::wcout << "Memory allocation and copying host->device finished" << std::endl;

	//3. ����������
	oddevensort(d_arr, arr_size);

	cudaStatus = hipGetLastError();
	if (cudaErrorOccured(cudaStatus)) return 1;

	cudaStatus = hipDeviceSynchronize();
	if (cudaErrorOccured(cudaStatus)) return 1;


	//4. ����������� ������� �������
	cudaStatus = hipMemcpy(arr, d_arr, arr_size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaErrorOccured(cudaStatus)) return 1;

	cudaStatus = hipFree(d_arr);
	if (cudaErrorOccured(cudaStatus)) return 1;

	cudaStatus = hipDeviceReset();;
	if (cudaErrorOccured(cudaStatus)) return 1;

	if (DEBUG_LEVEL >= 1) std::wcout << "Copying device->host and memory releasing finished" << std::endl;

	//5. ����� �������
	outputArray(arr, arr_size);
	delete[] arr;
	if (DEBUG_LEVEL >= 1) std::wcout << "Array output finished" << std::endl;
	
	if (DEBUG_LEVEL >= 1) {
		std::wcout << "Program finished" << std::endl;
	}

	if (DEBUG_LEVEL >= 2) pause();
    return 0;
}

